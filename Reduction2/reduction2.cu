#include "hip/hip_runtime.h"
// Kod do wersji 2 zadania.

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

// for srand( time( NULL ) )
#include <ctime>

/**
 * CUDA Kernel Device code
 Brak branch divergence! Warpy ktore policzyly swoja sume nie wykonuja ifa i sie koncza.
 nie ubiegaja sie o procesor!

 Sa konfikty ?

 */
__global__ void reduction(float *i_data, float *o_data, int numElements)
{
	extern __shared__ float sdata[];
	// Kazdy watek laduje jeden element z pamieci globalnej to pamieci wspoldzielonej
	unsigned int thId = threadIdx.x;							//ID w obrebie bloku
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;		//globalne id watku 
	sdata[thId] = 0;
	if (i < numElements)
		sdata[thId] = i_data[i];
	__syncthreads();

	// Redukcja w pamieci wspoldzielonej
	for (unsigned int s = 1; s < blockDim.x; s *= 2){
		int index = 2 * s * thId;

		if (index < blockDim.x){
			sdata[index] += sdata[index + s];				
		}												
		__syncthreads();
	}

	//zapis wyniku tego bloku do globalnej pamieci
	if (thId == 0)
		o_data[blockIdx.x] = sdata[0];
}

int main(void)
{
	hipError_t err = hipSuccess;

	int numElements = 75000000;
	
	   
	size_t size = numElements * sizeof(float);
	printf("[Vector reduction of %d elements]\n", numElements);

	//Determine amount of blocks and threads per block
	int threadsPerBlock = 512;
	int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

	size_t o_size = blocksPerGrid * sizeof(float);

	// Allocate the host vectors
	float *h_input = (float *)malloc(size);

	float h_output = 0;

	if (h_input == NULL)
	{
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

	// Initialize the host vector
	double checkSum = 0.0;
	for (int i = 0; i < numElements; ++i)
	{
		h_input[i] = 1;
		checkSum += h_input[i];
	}

	// Allocate the device input vector
	float *d_input = NULL;
	err = hipMalloc((void **)&d_input, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device input vector (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Allocate the device output vector
	float *d_output = NULL;
	err = hipMalloc((void **)&d_output, o_size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device output vector (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	printf("Copy input data from the host memory to the CUDA device\n");
	err = hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	bool turn = true;

	// Allocate CUDA events that we'll use for timing
	hipEvent_t start;
	err = hipEventCreate(&start);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	hipEvent_t stop;
	err = hipEventCreate(&stop);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Record the start event
	err = hipEventRecord(start, NULL);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	// Launch the reduction CUDA Kernel
	while (true){

		if (turn){

			reduction << <blocksPerGrid, threadsPerBlock, threadsPerBlock*sizeof(int) >> >(d_input, d_output, numElements);
			turn = false;
		}
		else{

			reduction << <blocksPerGrid, threadsPerBlock, threadsPerBlock*sizeof(int) >> >(d_output, d_input, numElements);
			turn = true;
		}

		err = hipGetLastError();
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to launch reduction kernel (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		if (blocksPerGrid == 1) break;

		numElements = blocksPerGrid;
		blocksPerGrid = ceil((double)blocksPerGrid / threadsPerBlock);

	}

	// Record the stop event
	err = hipEventRecord(stop, NULL);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Wait for the stop event to complete
	err = hipEventSynchronize(stop);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	float msecTotal = 0.0f;
	err = hipEventElapsedTime(&msecTotal, start, stop);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}



	// Sychronize threads ?
	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching kernel!\n", err);
		exit(EXIT_FAILURE);
	}


	// Copy results from device to host
	if (turn)
		err = hipMemcpy(&h_output, &d_input[0], sizeof(float), hipMemcpyDeviceToHost);
	else
		err = hipMemcpy(&h_output, &d_output[0], sizeof(float), hipMemcpyDeviceToHost);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy output vector from device to host (error code %s)!\n", hipGetErrorString(err));
		printf("turn = %d\n numElem = %d\n", turn, numElements);
		exit(EXIT_FAILURE);
	}

	// Verify that the result vector is correct

	if (h_output != checkSum)
	{
		fprintf(stderr, "Result verification failed! host result: %d !=  device result: %d\n", checkSum, h_output);
		exit(EXIT_FAILURE);
	}

	printf("Test PASSED\nTime: %f", msecTotal);

	// Free device global memory
	err = hipFree(d_input);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device input vector (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipFree(d_output);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device output vector (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Free host memory
	free(h_input);

	err = hipDeviceReset();

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	printf("Done\n");
	return 0;
}