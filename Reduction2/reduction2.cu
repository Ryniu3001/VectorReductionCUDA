#include "hip/hip_runtime.h"
// Kod do wersji 2 zadania.

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

// for srand( time( NULL ) )
#include <ctime>

/**
 * CUDA Kernel Device code
 Brak branch divergence! Warpy ktore policzyly swoja sume nie wykonuja ifa i sie koncza.
 nie ubiegaja sie o procesor!

 Sa konfikty ?

 */
__global__ void reduction(int *i_data, int *o_data, int numElements)
{
	extern __shared__ int sdata[];
	// Kazdy watek laduje jeden element z pamieci globalnej to pamieci wspoldzielonej
	unsigned int thId = threadIdx.x;							//ID w obrebie bloku ?
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;		//globalne id watku ??
	sdata[thId] = 0;
	if (i < numElements)
		sdata[thId] = i_data[i];
	__syncthreads();

	// Redukcja w pamieci wspoldzielonej
	for (unsigned int s = 1; s < blockDim.x; s *= 2){
		int index = 2 * s * thId;

		if (index < blockDim.x){
			sdata[index] += sdata[index + s];				
		}												
		__syncthreads();
	}

	//zapis wyniku tego bloku do globalnej pamieci
	if (thId == 0)
		o_data[blockIdx.x] = sdata[0];
}

int main(void)
{
	hipError_t err = hipSuccess;

	int numElements = 50000;
	size_t size = numElements * sizeof(int);
	printf("[Vector reduction of %d elements]\n", numElements);

	//Determine amount of blocks and threads per block
	int threadsPerBlock = 256;
	int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

	size_t o_size = blocksPerGrid * sizeof(int);

	// Allocate the host vectors
	int *h_input = (int *)malloc(size);

	int h_output = 0;

	if (h_input == NULL)
	{
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

	// Initialize the host vector
	int checkSum = 0;
	for (int i = 0; i < numElements; ++i)
	{
		h_input[i] = 1;
		checkSum += h_input[i];
	}

	// Allocate the device input vector
	int *d_input = NULL;
	err = hipMalloc((void **)&d_input, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device input vector (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Allocate the device output vector
	int *d_output = NULL;
	err = hipMalloc((void **)&d_output, o_size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device output vector (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	printf("Copy input data from the host memory to the CUDA device\n");
	err = hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Launch the reduction CUDA Kernel
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	bool turn = true;

	while (true){

		if (turn){

			reduction << <blocksPerGrid, threadsPerBlock, threadsPerBlock*sizeof(int) >> >(d_input, d_output, numElements);
			turn = false;
		}
		else{

			reduction << <blocksPerGrid, threadsPerBlock, threadsPerBlock*sizeof(int) >> >(d_output, d_input, numElements);
			turn = true;
		}

		err = hipGetLastError();
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to launch reduction kernel (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		if (blocksPerGrid == 1) break;

		numElements = blocksPerGrid;
		blocksPerGrid = ceil((double)blocksPerGrid / threadsPerBlock);

	}

	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching kernel!\n", err);
		exit(EXIT_FAILURE);
	}

	if (turn)
		err = hipMemcpy(&h_output, &d_input[0], sizeof(int), hipMemcpyDeviceToHost);
	else
		err = hipMemcpy(&h_output, &d_output[0], sizeof(int), hipMemcpyDeviceToHost);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy output vector from device to host (error code %s)!\n", hipGetErrorString(err));
		printf("turn = %d\n numElem = %d\n", turn, numElements);
		exit(EXIT_FAILURE);
	}

	// Verify that the result vector is correct

	if (h_output != checkSum)
	{
		fprintf(stderr, "Result verification failed! host result: %d !=  device result: %d\n", checkSum, h_output);
		exit(EXIT_FAILURE);
	}

	printf("Test PASSED\n");

	// Free device global memory
	err = hipFree(d_input);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device input vector (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipFree(d_output);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device output vector (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Free host memory
	free(h_input);

	err = hipDeviceReset();

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	printf("Done\n");
	return 0;
}